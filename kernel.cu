#include "hip/hip_runtime.h"
#define EIGEN_USE_GPU
#include <Eigen/Dense>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <thrust/device_vector.h>
#include <vector>
#include<chrono>
using namespace Eigen;
using namespace std;

__global__
void Moving(int row,double* mp_gpu, double* mpd_gpu, double moving_point_step, double* ep_gpu, int* in_edge, 
    int* moving_point_cover_gpu, int* next_cover_gpu, int* before_cover_gpu, int* f, double* v,int rowv, 
    int* neighbor_gpu){
	Eigen::Map<Eigen::MatrixXd> moving_point(mp_gpu, row, 3);
	Eigen::Map<Eigen::MatrixXd> moving_point_direct(mpd_gpu, row, 3);
	Eigen::Map<Eigen::MatrixXd> edge_point(ep_gpu, row, 3);
    Eigen::Map<Eigen::MatrixXi> moving_point_cover(moving_point_cover_gpu, row, 1);
	Eigen::Map<Eigen::MatrixXi> before_cover(before_cover_gpu, row, 1);
	Eigen::Map<Eigen::MatrixXi> next_cover(next_cover_gpu, row, 1);
	Eigen::Map<Eigen::MatrixXi> F(f, row, 3);
	Eigen::Map<Eigen::MatrixXd> V(v, rowv, 3);
    Eigen::Map<Eigen::MatrixXi> neighbor(neighbor_gpu, row, 3);
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = gridDim.x * blockDim.x;
	for (int j = i;  j < row; j += stride) {
		if (j  < row) {
			if (moving_point.row(j) == RowVector3d(0, 0, 0)) {
				continue;
			}
			////判断是否到边
			if (in_edge[j] == 1) {
                int end = 0;
                in_edge[j] = 0;
                int cover = moving_point_cover(j, 0);
                int a = F(cover, 0);
                int b = F(cover, 1);
                int c = F(cover, 2);
                //BC
                double t1 = (moving_point(j, 1) - V(b, 1)) * (V(c, 2) - moving_point(j, 2)) - (moving_point(j, 2) - V(b, 2)) * (V(c, 1) - moving_point(j, 1));
                double t2 = (V(c, 1) - V(b, 1)) * moving_point_direct(j, 2) - (V(c, 2) - V(b, 2)) * moving_point_direct(j, 1);
                double t = t1 / t2;
                if (t > 0) {
                    RowVector3d point = moving_point.row(j) + t * moving_point_direct.row(j);
                    Eigen::RowVector3d v1 = V.row(b) - point;
                    Eigen::RowVector3d v2 = V.row(c) - point;

                    if (v1.dot(v2) < 0 && neighbor(cover, 2) != before_cover(j, 0)) {
                        before_cover(j, 0) = cover;
                        edge_point.row(j) = point;
                        next_cover(j, 0) = neighbor(cover, 2);
                        end = 1;
                    }

                }
                //AB
                t1 = (moving_point(j, 1) - V(a, 1)) * (V(b, 2) - moving_point(j, 2)) - (moving_point(j, 2) - V(a, 2)) * (V(b, 1) - moving_point(j, 1));
                t2 = (V(b, 1) - V(a, 1)) * moving_point_direct(j, 2) - (V(b, 2) - V(a, 2)) * moving_point_direct(j, 1);
                t = t1 / t2;
                if (t > 0 && end==0) {
                    RowVector3d point = moving_point.row(j) + t * moving_point_direct.row(j);
                    Eigen::RowVector3d v1 = V.row(a) - point;
                    Eigen::RowVector3d v2 = V.row(b) - point;
                    if (v1.dot(v2) < 0 && neighbor(cover, 0) != before_cover(j, 0)) {
                        before_cover(j, 0) = cover;
                        edge_point.row(j) = point;
                        next_cover(j, 0) = neighbor(cover, 0);
                        end = 1;
                    }

                }
                //AC
                t1 = (moving_point(j, 1) - V(a, 1)) * (V(c, 2) - moving_point(j, 2)) - (moving_point(j, 2) - V(a, 2)) * (V(c, 1) - moving_point(j, 1));
                t2 = (V(c, 1) - V(a, 1)) * moving_point_direct(j, 2) - (V(c, 2) - V(a, 2)) * moving_point_direct(j, 1);
                t = t1 / t2;
                if (t > 0 && end == 0) {
                    RowVector3d point = moving_point.row(j) + t * moving_point_direct.row(j);
                    Eigen::RowVector3d v1 = V.row(a) - point;
                    Eigen::RowVector3d v2 = V.row(c) - point;
                    if (v1.dot(v2) < 0 && neighbor(cover, 1) != before_cover(j, 0)) {
                        before_cover(j, 0) = cover;
                        edge_point.row(j) = point;
                        next_cover(j, 0) = neighbor(cover, 1);
                        end = 1;
                    }

                }
                if (end == 0) {
                    moving_point_direct.row(j) = RowVector3d(0, 0, 0);
                    moving_point.row(j) = RowVector3d(0, 0, 0);
                    edge_point.row(j) = RowVector3d(0, 0, 0);
                }
			}
			//计算现在的点的位置
			moving_point(j, 0) = moving_point(j, 0) + moving_point_direct(j, 0) * moving_point_step;
			moving_point(j, 1) = moving_point(j, 1) + moving_point_direct(j, 1) * moving_point_step;
			moving_point(j, 2) = moving_point(j, 2) + moving_point_direct(j, 2) * moving_point_step;
		}
	}
}


extern "C" void moving_point_gpu(int row, double* mp_gpu, double* mpd_gpu, double moving_point_step,
    double* ep_gpu, int* in_edge, int* moving_point_cover_gpu, int* next_cover_gpu, int* before_cover_gpu,
    int* f, double* v, int rowv, int* neighbor_gpu){
    Moving << <2, 256 >> > (row, mp_gpu, mpd_gpu, moving_point_step, 
        ep_gpu, in_edge, moving_point_cover_gpu, next_cover_gpu, 
        before_cover_gpu,f,v,rowv, neighbor_gpu);	
    hipDeviceSynchronize();
	
}